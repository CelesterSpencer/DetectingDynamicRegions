#include "hip/hip_runtime.h"
//#include "scaleByDepth.h"
#include "hip/hip_runtime.h"
#include "opencv2/gpu/gpu.hpp"

extern "C" {
    void scaleDepth();
}

__global__ void scaleByDepth(float *flowX, float *flowY, float *depth, int size) {

    // get position within opticalflowfield
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // get value for calculated position
    if (i >= size) return; // check bounds

    // scalar product
    float vectorlength = sqrt(flowX[i] * flowX[i] + flowY[i] * flowY[i]) ;

    printf("ich bin thread %d \n", i);
}

void scaleDepth(cv::gpu::GpuMat flowX, cv::gpu::GpuMat flowY, cv::gpu::GpuMat depth, cv::gpu::GpuMat vec3D, int threadsize) {
    int size = flowX.rows * flowX.cols;
    int blocksize = size / threadsize;
    float *dataX = (float*)flowX.data;
    float *dataY = (float*)flowY.data;
    float *dataD = (float*)depth.data;

    float *d__dataX;
    float *d__dataY;
    float *d__dataD;
    hipMalloc((float**)&d__dataX, sizeof(float) * size);
    hipMalloc((float**)&d__dataY, sizeof(float) * size);
    hipMalloc((float**)&d__dataD, sizeof(float) * size);
    hipMemcpy(d__dataX, dataX, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d__dataY, dataY, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d__dataD, dataD, sizeof(float) * size, hipMemcpyHostToDevice);

    scaleByDepth<<<blocksize, threadsize>>>(d__dataX, d__dataY, d__dataD, size);
    hipDeviceSynchronize();

    hipMemcpy(vec3D.data, d__dataD, sizeof(float) * size, hipMemcpyDeviceToHost);

    hipFree(d__dataX);
    hipFree(d__dataY);
    hipFree(d__dataD);
}
